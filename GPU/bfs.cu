#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

using namespace std;

#define cudaCheck(error) \
  if (error != hipSuccess) { \
    printf("Fatal error: %s at %s:%d\n", \
      hipGetErrorString(error), \
      __FILE__, __LINE__); \
    exit(1); \
  }

__global__ void cudawbfs(int *distance, unsigned int *row_ptr, int *col_ind, int nov, int *improvement, int level)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int localImprovement = 0;

  if(tid < nov && distance[tid] == level) {
    for(int e = row_ptr[tid]; e < row_ptr[tid + 1]; e++){
      int adj = col_ind[e];
      if(distance[adj] < 0){
        distance[adj] = level + 1;
        localImprovement = 1;
      }
    }
  }
  
  if(localImprovement) {
    (*improvement) = localImprovement;
  }
}

void wbfs(unsigned int * row_ptr, int * col_ind, int * distance, int nov, int * d_distance, unsigned int * d_row_ptr, int * d_col_ind){
  //initializations
  int size_of_rowptr = (nov + 1) * sizeof(int);
  int size_of_colind = row_ptr[nov] * sizeof(int);
  int *d_improvement, *d_nov, *d_level;

  //memory allocations
  cudaCheck(hipMalloc((void**) &d_improvement, sizeof(int)));
  cudaCheck(hipMalloc((void**) &d_nov, sizeof(int)));
  cudaCheck(hipMalloc((void**) &d_level, sizeof(int)));
  cudaCheck(hipMalloc((void**) &d_row_ptr, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_distance, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_col_ind, size_of_colind));

  //memory copies
  cudaCheck(hipMemcpy(d_distance, distance, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_row_ptr, row_ptr, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_col_ind, col_ind, size_of_colind, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_nov, &nov, sizeof(int), hipMemcpyHostToDevice));


  //start time
  hipEvent_t start;
  hipEvent_t stop;

  cudaCheck(hipEventCreate(&start));
  cudaCheck(hipEventCreate(&stop));
  cudaCheck(hipEventRecord(start, 0));

  int *improvement = new int;
  int level = 1;
  do{
     (*improvement) = 0;
     cudaCheck(hipMemcpy(d_improvement, improvement, sizeof(int), hipMemcpyHostToDevice));
     cudawbfs<<<(nov + 1023) / 1024, 1024>>>(d_distance, d_row_ptr, d_col_ind, nov, d_improvement, level);
     cudaCheck(hipMemcpy(improvement, d_improvement, sizeof(int), hipMemcpyDeviceToHost));
     level++;
  } while((*improvement) == 1);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed;
  hipEventElapsedTime(&elapsed, start, stop);

  //take value again
  cudaCheck(hipMemcpy(distance, d_distance, size_of_rowptr, hipMemcpyDeviceToHost));

  //deallocations
  cudaCheck(hipFree(d_row_ptr));
  cudaCheck(hipFree(d_distance));
  cudaCheck(hipFree(d_col_ind));

  printf("GPU WBFS time: %f s\n", elapsed / 1000);
}

__global__ void cudaqbfs(int *distance, unsigned int *row_ptr, int *col_ind, int *queue, int *nextQueue, int size, int *nextSize, int level) {
  int index, u, v, tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < size) {
    u = queue[tid];
    for(int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
      v = col_ind[e];
      if (distance[v] == -1) {
        distance[v] = level + 1;
        index = atomicAdd(nextSize, 1);
        nextQueue[index] = v;
      }
    }
  }
}

void qbfs(unsigned int *row_ptr, int *col_ind, int *distance, int nov, int source) {
  int srcNeigh = row_ptr[source + 1] - row_ptr[source];
  int *srcArr = new int[srcNeigh];
  int index = 0;
  for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) {
    if (distance[col_ind[i]] == 1) {
      srcArr[index++] = col_ind[i];
    }
  }
  
  int size_of_rowptr = (nov + 1) * sizeof(int);
  int size_of_colind = row_ptr[nov] * sizeof(int);
  unsigned int *d_row_ptr;
  int *d_col_ind, *d_distance, *d_queue, *d_nextQueue, *d_nextSize;

  cudaCheck(hipMalloc((void**) &d_row_ptr, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_col_ind, size_of_colind));
  cudaCheck(hipMalloc((void**) &d_distance, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_queue, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_nextQueue, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_nextSize, sizeof(int)));

  cudaCheck(hipMemcpy(d_distance, distance, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_row_ptr, row_ptr, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_col_ind, col_ind, size_of_colind, hipMemcpyHostToDevice));
  //cudaCheck(cudaMemcpy(d_queue, &source, sizeof(int), cudaMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));

	hipEvent_t start;
 	hipEvent_t stop;
 	cudaCheck(hipEventCreate(&start));
 	cudaCheck(hipEventCreate(&stop));
 	cudaCheck(hipEventRecord(start, 0));

  int size = srcNeigh;
  int *nextSize = new int;
  *nextSize = 0;
  int level = 1;
  do {
    cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));
    cudaqbfs<<<(size + 1023) / 1024, 1024>>>(d_distance, d_row_ptr, d_col_ind, d_queue, d_nextQueue, size, d_nextSize, level);
    cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));
    level++;
    size = *nextSize;
    *nextSize = 0;
    swap(d_queue, d_nextQueue);
  } while(size > 0);

	hipEventRecord(stop, 0);
 	hipEventSynchronize(stop);
 	float elapsed;
 	hipEventElapsedTime(&elapsed, start, stop);

  cudaCheck(hipMemcpy(distance, d_distance, size_of_rowptr, hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_row_ptr));
  cudaCheck(hipFree(d_col_ind));
  cudaCheck(hipFree(d_distance));
  cudaCheck(hipFree(d_queue));
  cudaCheck(hipFree(d_nextQueue));
  cudaCheck(hipFree(d_nextSize));

  printf("GPU QBFS time: %f s\n", elapsed / 1000);
}


__global__ void cudatdwbfs(int *distance, unsigned int *row_ptr, int *col_ind, int nov, int level, int *mf) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < nov && distance[tid] == level) {     
     for(int e = row_ptr[tid]; e < row_ptr[tid + 1]; e++) {
       int adj = col_ind[e];
       if(distance[adj] < 0) {
         atomicAdd(mf, -distance[adj]);
         distance[adj] = level + 1;
       }
     }
   }
}

__global__ void cudatdqbfs(int *distance, unsigned int *row_ptr, int *col_ind, int *queue, int *nextQueue, int size, int *nextSize, int level, int *mf) {
  int index, u, v, tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < size) {
    u = queue[tid];
    for(int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
      v = col_ind[e];
      if (distance[v] < 0) {
        index = atomicAdd(nextSize, 1);
        atomicAdd(mf, -distance[v]);
        distance[v] = level + 1;
        nextQueue[index] = v;
      }
    }
  }
}

__global__ void cudabuwbfs(int *distance, unsigned int *row_ptr_inv, int *col_ind_inv, int nov, int level, int *mf) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < nov && distance[tid] < 0) {
    for(int e = row_ptr_inv[tid]; e < row_ptr_inv[tid + 1]; e++) {
      int adj = col_ind_inv[e];
      if(distance[adj] == level) {
        atomicAdd(mf, -distance[tid]);
        distance[tid] = level + 1;
        break;
      }
    }
  }
}

__global__ void cudabuqbfs(int *distance, unsigned int *row_ptr_inv, int *col_ind_inv, int nov, int level, int *nextSize, int *mf) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < nov && distance[tid] < 0) {
    for(int e = row_ptr_inv[tid]; e < row_ptr_inv[tid + 1]; e++) {
      int adj = col_ind_inv[e];
      if(distance[adj] == level) {
        atomicAdd(mf, -distance[tid]);
        atomicAdd(nextSize, 1);
        distance[tid] = level + 1;
        break;
      }
    }
  }
}

void hybrid(unsigned int *row_ptr, unsigned int *row_ptr_inv, int *col_ind, int *col_ind_inv, int *distance, int nov, int source, double alpha) { // int init_mf?
	int size_of_rowptr = (nov + 1) * sizeof(int);
  int size_of_colind = row_ptr[nov] * sizeof(int);
  int *improvement = new int;
  unsigned int *d_row_ptr, *d_row_ptr_inv;
  int *d_col_ind, *d_col_ind_inv, *d_distance, *d_mf;

  cudaCheck(hipMalloc((void**) &d_row_ptr, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_row_ptr_inv, size_of_rowptr));
	cudaCheck(hipMalloc((void**) &d_col_ind, size_of_colind));
	cudaCheck(hipMalloc((void**) &d_col_ind_inv, size_of_colind));
  cudaCheck(hipMalloc((void**) &d_distance, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_mf, sizeof(int)));

  cudaCheck(hipMemcpy(d_distance, distance, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_row_ptr, row_ptr, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_row_ptr_inv, row_ptr_inv, size_of_rowptr, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_col_ind, col_ind, size_of_colind, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_col_ind_inv, col_ind_inv, size_of_colind, hipMemcpyHostToDevice));


  int srcNeigh = row_ptr[source + 1] - row_ptr[source];
  int *srcArr = new int[srcNeigh];
  int index = 0;
  for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) {
    if (distance[col_ind[i]] == 1) {
      srcArr[index++] = col_ind[i];
    }
  }

  int *d_queue, *d_nextQueue, *d_nextSize;
  cudaCheck(hipMalloc((void**) &d_queue, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_nextQueue, size_of_rowptr));
  cudaCheck(hipMalloc((void**) &d_nextSize, sizeof(int)));

  //cudaCheck(cudaMemcpy(d_queue, &source, sizeof(int), cudaMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));

  int mf = row_ptr[source + 1] - row_ptr[source]; // number of traversed edges
  int mu = row_ptr[nov];                          // total number of edges
  int prev_mf = -1;
  int level = 1;

  int size = srcNeigh;
  int *nextSize = new int;
  *nextSize = 0;

	hipEvent_t start;
 	hipEvent_t stop;
 	cudaCheck(hipEventCreate(&start));
 	cudaCheck(hipEventCreate(&stop));
  cudaCheck(hipEventRecord(start, 0));

  while (mf != prev_mf) {
    prev_mf = mf;
    if (mf > mu / alpha) {
      cudaCheck(hipMemcpy(d_mf, &mf, sizeof(int), hipMemcpyHostToDevice));
      cudabuwbfs<<<(nov + 1023) / 1024, 1024>>>(d_distance, d_row_ptr_inv, d_col_ind_inv, nov, level, d_mf);
      cudaCheck(hipMemcpy(&mf, d_mf, sizeof(int), hipMemcpyDeviceToHost));
    }
    else {
      cudaCheck(hipMemcpy(d_mf, &mf, sizeof(int), hipMemcpyHostToDevice));
      cudatdwbfs<<<(nov + 1023) / 1024, 1024>>>(d_distance, d_row_ptr, d_col_ind, nov, level, d_mf);      
      cudaCheck(hipMemcpy(&mf, d_mf, sizeof(int), hipMemcpyDeviceToHost));
    }
    level++;
  }

  /*while (mf != prev_mf) {
    prev_mf = mf;
    if (mf > mu / alpha) {
      cudaCheck(cudaMemcpy(d_mf, &mf, sizeof(int), cudaMemcpyHostToDevice));
      cudaCheck(cudaMemcpy(d_nextSize, nextSize, sizeof(int), cudaMemcpyHostToDevice));
      cudabuqbfs<<<(nov + 1023) / 1024, 1024>>>(d_distance, d_row_ptr_inv, d_col_ind_inv, nov, level, d_nextSize, d_mf);
      cudaCheck(cudaMemcpy(&mf, d_mf, sizeof(int), cudaMemcpyDeviceToHost));
      cudaCheck(cudaMemcpy(nextSize, d_nextSize, sizeof(int), cudaMemcpyDeviceToHost));
    }
    else {
      cudaCheck(cudaMemcpy(d_mf, &mf, sizeof(int), cudaMemcpyHostToDevice));
      cudaCheck(cudaMemcpy(d_nextSize, nextSize, sizeof(int), cudaMemcpyHostToDevice));
      cudatdqbfs<<<(size + 1023) / 1024, 1024>>>(d_distance, d_row_ptr, d_col_ind, d_queue, d_nextQueue, size, d_nextSize, level, d_mf);      
      cudaCheck(cudaMemcpy(&mf, d_mf, sizeof(int), cudaMemcpyDeviceToHost));
      cudaCheck(cudaMemcpy(nextSize, d_nextSize, sizeof(int), cudaMemcpyDeviceToHost));
    }
    level++;     
    size = *nextSize;
    *nextSize = 0;
    swap(d_queue, d_nextQueue);
  }*/

	hipEventRecord(stop, 0);
 	hipEventSynchronize(stop);
 	float elapsed;
 	hipEventElapsedTime(&elapsed, start, stop);

  cudaCheck(hipMemcpy(distance, d_distance, size_of_rowptr, hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_row_ptr));
  cudaCheck(hipFree(d_row_ptr_inv));
  cudaCheck(hipFree(d_col_ind));
  cudaCheck(hipFree(d_col_ind_inv));
  cudaCheck(hipFree(d_distance));
  cudaCheck(hipFree(d_mf));
  cudaCheck(hipFree(d_queue));
  cudaCheck(hipFree(d_nextQueue));
  cudaCheck(hipFree(d_nextSize));

  printf("GPU Hybrid time: %f s\n", elapsed / 1000);

}